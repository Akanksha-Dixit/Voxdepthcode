#include "hip/hip_runtime.h"
#include "unifiedSurface.h"



#define CX 322.5f
#define CY 181.4f
#define FXY 325.5f
#define KERSIZE 5
#define USMAX 65535


int szMat;
dim3 block1,grid1;
float* inputMat,* pointCloud,*transform,*outputMat;
int imgHit,imgWid;
void allocUnifiedMem(int imgW,int imgH)
{
    imgHit = imgH;
    imgWid = imgW;
    szMat = imgH*imgW;
    block1.x = 32;
    block1.y = 32;
    grid1.x = (imgH + block1.x-1)/block1.x;
    grid1.y = (imgW + block1.y-1)/block1.y;
    
    hipMallocManaged(&inputMat,szMat*sizeof(float));
    hipMallocManaged(&outputMat,szMat*sizeof(float));
    hipMallocManaged(&pointCloud,3*szMat*sizeof(float));
    hipMallocManaged(&transform,16*sizeof(float));
}
void setUnifiedMem(float* inputMap)
{
    std::memcpy(inputMat,inputMap,szMat*sizeof(float));
}
void setTrans(cv::Mat &Rt)
{
    std::cout<<transform[0]<<" "<<transform[1]<<" "<<transform[2]<<" "<<transform[3]<<std::endl;
    std::cout<<transform[4]<<" "<<transform[5]<<" "<<transform[6]<<" "<<transform[7]<<std::endl;
    std::cout<<transform[8]<<" "<<transform[9]<<" "<<transform[10]<<" "<<transform[11]<<std::endl;
}
__global__ void transformPCD(float* pointCloud,float* transform,int imgW,int imgH)
{
    int xIndex = blockIdx.x * blockDim.x + threadIdx.x; //image x index or column number
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y; //image y index of row number
    if(xIndex<imgH && yIndex<imgW)
    {
        
        float x = pointCloud[3*(xIndex*imgW + yIndex)];
        float y = pointCloud[3*(xIndex*imgW + yIndex) + 1];
        float z = pointCloud[3*(xIndex*imgW + yIndex) + 2];
        // printf("Before (%d,%d):(%f,%f,%f)\n",xIndex,yIndex,pointCloud[3*(xIndex*imgW + yIndex)],pointCloud[3*(xIndex*imgW + yIndex)+1],z);

        pointCloud[3*(xIndex*imgW + yIndex)] = x*transform[0] + y*transform[1] + z*transform[2] + transform[3];
        pointCloud[3*(xIndex*imgW + yIndex) + 1] = x*transform[4] + y*transform[5] + z*transform[6] + transform[7];
        pointCloud[3*(xIndex*imgW + yIndex) + 2] = x*transform[8] + y*transform[9] + z*transform[10] + transform[11];

        printf("After (%d,%d):(%f,%f,%f,%f)-(%f,%f,%f):(%f,%f,%f)\n",xIndex,yIndex,transform[0],transform[1],transform[2],transform[3],x,y,z,pointCloud[3*(xIndex*imgW + yIndex)],pointCloud[3*(xIndex*imgW + yIndex)+1],pointCloud[3*(xIndex*imgW + yIndex)+2]);
    }

}
__global__ void kernelToSurfaceUnified(float* inputMat,float *outputMat,float *pointCloud,int imgW,int imgH,float cutoff)
{
    int xIndex = blockIdx.x * blockDim.x + threadIdx.x; //image x index or column number
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y; //image y index of row number
    if(xIndex<imgH && yIndex<imgW)
    {
        float z = inputMat[xIndex*imgW + yIndex];
        if(z<=cutoff)
        {
            pointCloud[3*(xIndex*imgW + yIndex)] = 0.0f;
            pointCloud[3*(xIndex*imgW + yIndex) + 1] = 0.0f;
            pointCloud[3*(xIndex*imgW + yIndex) + 2] = 0.0f;
            outputMat[xIndex*imgW + yIndex] = (float)USMAX;
        }
        else if(pointCloud[3*(xIndex*imgW + yIndex) + 2]>z || pointCloud[3*(xIndex*imgW + yIndex) + 2]==0.0f)
        {
            pointCloud[3*(xIndex*imgW + yIndex)] = (((float)xIndex-CX)*z)/FXY;
            pointCloud[3*(xIndex*imgW + yIndex) + 1] = (((float)yIndex-CY)*z)/FXY;
            pointCloud[3*(xIndex*imgW + yIndex) + 2] = z;
            outputMat[xIndex*imgW + yIndex] = z*1000;
            // printf("(%d,%d):(%f,%f,%f)\n",xIndex,yIndex,pointCloud[3*(xIndex*imgW + yIndex)],pointCloud[3*(xIndex*imgW + yIndex)+1],z);
        }
        
    }
}

__global__ void medFilter(float* inputMat,float *outputMat,int imgW,int imgH,int kerSize)
{
    int xIndex = blockIdx.x * blockDim.x + threadIdx.x; //image x index or column number
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y; //image y index of row number
    int halfKer = kerSize/2;
    int cnt = kerSize*kerSize;
    if(xIndex<(imgH-halfKer-1) && yIndex<(imgW-halfKer-1) && xIndex>halfKer+1 && yIndex>halfKer+1)
    {
        float sorted[KERSIZE*KERSIZE];
        int initCnt = 0;
        //store vals
        for(int i=xIndex-halfKer;i<=xIndex+halfKer;i++)
        {
            for(int j=yIndex-halfKer;j<=yIndex+halfKer;j++)
            {
                sorted[initCnt] = inputMat[i*imgW + j];
                initCnt+=1;
            }
             
        }
        
        //sort vals
        for(int i=0;i<cnt;i++)
        {
            for(int j=0;j<cnt;j++)
            {
                if(sorted[j]>sorted[j+1])
                {
                    float temp = sorted[j];
                    sorted[j] = sorted[j+1];
                    sorted[j+1] = temp;
                }
            }
        }

        //replace value
        outputMat[xIndex*imgW + yIndex] = sorted[cnt/2];

    }
}

void destroyUnifiedMem()
{
    hipFree(inputMat);
    hipFree(outputMat);
    hipFree(pointCloud);
    hipFree(transform);
}
void toSurfaceCallerUnified()
{

    kernelToSurfaceUnified<<<grid1,block1>>>(inputMat,outputMat,pointCloud,imgWid,imgHit,5.0f);
    hipDeviceSynchronize();
    
}
void transformPCDCaller()
{
    transformPCD<<<grid1,block1>>>(pointCloud,transform,imgWid,imgHit);
    hipDeviceSynchronize();
}
void copyPC(float *pc)
{
    std::memcpy(pc,pointCloud,3*szMat*sizeof(float));
}
void copyDepthImg(short* depthImg)
{

    medFilter<<<grid1,block1>>>(outputMat,inputMat,imgWid,imgHit,KERSIZE);
    hipDeviceSynchronize();

    for(int i=0;i<szMat;i++)
    {
        depthImg[i] = std::floor(outputMat[i]);
    }

    // for(int i=0;i<szMat;i++)
    // {
    //     float z = pointCloud[3*i+2];
    //     float x = pointCloud[3*i];
    //     float y = pointCloud[3*i+1];
    //     // std::cout<<z<<"\n";
    //     if(z==0.0f)
    //         continue;
    //     int d = std::floor(z*1000);
    //     int u = std::round((x*FXY)/z + CX);
    //     int v = std::round((y*FXY)/z + CY);
    //     if(u<imgHit and v<imgWid)
    //     {
    //         depthImg[u*imgWid + v]=d;
    //     }
    // }
}