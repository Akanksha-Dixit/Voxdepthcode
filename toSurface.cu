#include "hip/hip_runtime.h"
#include "toSurface.h"
#include <iostream>
#include<chrono>
#include <cmath>

#define CX 322.5f
#define CY 181.4f
#define FXY 325.5f

__global__ void kernelToSurface(float* inputMat,float *pointCloud,int imgW,int imgH)
{
    int xIndex = blockIdx.x * blockDim.x + threadIdx.x; //image x index or column number
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y; //image y index of row number
    if(xIndex<imgH && yIndex<imgW)
    {
        float z = inputMat[xIndex*imgW + yIndex];
        if(z==0.0f)
        {
            pointCloud[3*(xIndex*imgW + yIndex)] = 0.0f;
            pointCloud[3*(xIndex*imgW + yIndex) + 1] = 0.0f;
            pointCloud[3*(xIndex*imgW + yIndex) + 2] = 0.0f;
        }
        else
        {
            pointCloud[3*(xIndex*imgW + yIndex)] = (((float)xIndex-CX)*z)/FXY;
            pointCloud[3*(xIndex*imgW + yIndex) + 1] = (((float)yIndex-CY)*z)/FXY;
            pointCloud[3*(xIndex*imgW + yIndex) + 2] = z;
        }
        // printf("(%d,%d):(%f,%f,%f)\n",xIndex,yIndex,pointCloud[3*(xIndex*imgW + yIndex)],pointCloud[3*(xIndex*imgW + yIndex)+1],z);
    }
}

__global__ void testKernel(float* inputMat,float* pointCloud,int imgW,int imgH)
{
    int xIndex = blockIdx.x * blockDim.x + threadIdx.x; //image x index or column number
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y; //image y index of row number
    if(xIndex<imgH && yIndex<imgW)
    {
        inputMat[xIndex*imgW + yIndex] = 0.0f;
        // printf("%f",inputMat[xIndex*imgW + yIndex]);
        pointCloud[3*(xIndex*imgW + yIndex)] = 11.0f;
        pointCloud[3*(xIndex*imgW + yIndex) + 1] = 11.0f;
        pointCloud[3*(xIndex*imgW + yIndex) + 2] = 11.0f;
    }
}

void toImage(float *pointCloud,short *outImg,long long sz,int imgH,int imgW)
{
    for(int i=0;i<sz;i++)
    {
        float z = pointCloud[3*i+2];
        float x = pointCloud[3*i];
        float y = pointCloud[3*i+1];
        if(z==0.0f)
            continue;
        int d = std::floor(z*1000);
        int u = std::round((x*FXY)/z + CX);
        int v = std::round((y*FXY)/z + CY);
        if(u<imgH and v<imgW)
                outImg[u*imgW + v]=d;
    }

}

int sz;
float *d_inputMat,*d_pointCloud;
dim3 block,grid;

void setDeviceMem(int imgW,int imgH)
{
    std::cout<<"Memset\n";
    sz = imgH*imgW;
    hipMalloc(&d_inputMat,sz*sizeof(float));
    hipMalloc(&d_pointCloud,3*(sz)*sizeof(float));
    block.x = 32;
    block.y = 32;
    grid.x = (imgH + block.x-1)/block.x;
    grid.y = (imgW + block.y-1)/block.y;
}

void toSurfaceCaller(float* inputMat,float* pointCloud,int imgW,int imgH)
{
    hipMemcpy(d_inputMat, inputMat, sz * sizeof(float), hipMemcpyHostToDevice);
    kernelToSurface<<<grid,block>>>(d_inputMat,d_pointCloud,imgW,imgH);
    hipMemcpy(pointCloud,d_pointCloud,3*sz*sizeof(float),hipMemcpyDeviceToHost);
}


void destroyMem()
{
    hipFree(d_inputMat);
    hipFree(d_pointCloud);
}