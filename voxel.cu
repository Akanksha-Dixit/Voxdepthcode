#include "hip/hip_runtime.h"
#include "voxel.h"

//realsense
#define CX 322.5f
#define CY 181.4f
#define FXY 325.5f


//Midair
// #define CX 512.0f
// #define CY 512.0f
// #define FXY 512.0f

#define XYDIV 0.1f
#define ZDIV 0.065f
#define KERSIZE 5
#define USMAX 65535
#define VOXSIZE 1000
#define XYVOXMAX 100
#define ZVOXMAX 65
#define BUILDING true


int szMat;
dim3 block1,grid1;
dim3 blockVox,gridVox;
float* inputMat,* pointCloud,*transform,*outputMat;
char* voxelGrid;
int imgHit,imgWid;

////////////////////////////////////////////////////////
//
// KERNELS
//
////////////////////////////////////////////////////////
/*
__device__ static inline char atomicAdd(char* address, char val) {
    // offset, in bytes, of the char* address within the 32-bit address of the space that overlaps it
    size_t long_address_modulo = (size_t) address & 3;
    // the 32-bit address that overlaps the same memory
    auto* base_address = (unsigned int*) ((char*) address - long_address_modulo);
    // A 0x3210 selector in __byte_perm will simply select all four bytes in the first argument in the same order.
    // The "4" signifies the position where the first byte of the second argument will end up in the output.
    unsigned int selectors[] = {0x3214, 0x3240, 0x3410, 0x4210};
    // for selecting bytes within a 32-bit chunk that correspond to the char* address (relative to base_address)
    unsigned int selector = selectors[long_address_modulo];
    unsigned int long_old, long_assumed, long_val, replacement;

    long_old = *base_address;

    do {
        long_assumed = long_old;
        // replace bits in long_old that pertain to the char address with those from val
        long_val = __byte_perm(long_old, 0, long_address_modulo) + val;
        replacement = __byte_perm(long_old, long_val, selector);
        long_old = atomicCAS(base_address, long_assumed, replacement);
    } while (long_old != long_assumed);
    return __byte_perm(long_old, 0, long_address_modulo);
}
*/
__global__ void kernelToSurfaceUnified(float* inputMat,char* voxelGrid,float* outputMat,int imgW,int imgH)
{
    int xIndex = blockIdx.x * blockDim.x + threadIdx.x; //image x index or column number
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y; //image y index of row number
    if(xIndex<imgH && yIndex<imgW)
    {
        float z = inputMat[xIndex*imgW + yIndex];
        if(z!=0.0)
        {
            int xVox = (int)floor(((((float)xIndex-CX)*z)/FXY)/XYDIV) + 500;
            int yVox = (int)floor(((((float)yIndex-CY)*z)/FXY)/XYDIV) + 500;
            int zVox = (int)floor(z/ZDIV);
            
            if(xVox>=0 && yVox>=0 && zVox>=0 && xVox<VOXSIZE && yVox<VOXSIZE && zVox<VOXSIZE)
            {
                voxelGrid[xVox + yVox*VOXSIZE + zVox*VOXSIZE*VOXSIZE] = 1;
                if(BUILDING)
                {
                    float xCoor = ((float)(xVox-500)*XYDIV);
                    float yCoor = ((float)(yVox-500)*XYDIV);
                    int u = round(((xCoor)*FXY)/z + CX);
                    int v = round(((yCoor)*FXY)/z + CY);
                    float zNew = ((float)zVox)*ZDIV*1000;
                    if(u<imgH && v<imgW)
                    {
                        outputMat[u*imgW+v] = zNew;
                    }
                }
            }

        }
        
    }
}
__global__ void combineDepthKernel(float* inputMat,float* outputMat,int imgW,int imgH,float depthCutoff)
{
    int xIndex = blockIdx.x * blockDim.x + threadIdx.x; //image x index or column number
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y; //image y index of row number
    if(xIndex<imgH && yIndex<imgW)
    {
        
        if(inputMat[xIndex*imgW + yIndex]>=65.0f || inputMat[xIndex*imgW + yIndex]==0.0f)
        {
            inputMat[xIndex*imgW + yIndex] = outputMat[xIndex*imgW + yIndex];
        }
        if(inputMat[xIndex*imgW + yIndex]<=depthCutoff)
        {
            inputMat[xIndex*imgW + yIndex] = (float) USMAX;
        }

    }
}
__global__ void kernelSurfaceToImage(char* voxelGrid,float* outputMat,int imgW,int imgH)
{
    int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    int zIndex = blockIdx.z * blockDim.z + threadIdx.z;
    if(xIndex<1000 && yIndex<1000 && zIndex<1000)
    {
        //440,584,95
        
           
        float z = ((float)zIndex*ZDIV)*1000;
        // printf("%f\n",z);
        float xCoor = ((float)(xIndex-500)*XYDIV);
        float yCoor = ((float)(yIndex-500)*XYDIV);
        int u = round(((xCoor)*FXY)/z + CX);
        int v = round(((yCoor)*FXY)/z + CY);
        if(u<imgH && v<imgW )
        {
            if(outputMat[u*imgW+v]>z)
                outputMat[u*imgW+v] =z;
        }
        
    }
}

__global__ void medFilter(float* inputMat,float *outputMat,int imgW,int imgH,int kerSize)
{
    int xIndex = blockIdx.x * blockDim.x + threadIdx.x; //image x index or column number
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y; //image y index of row number
    int halfKer = kerSize/2;
    int cnt = kerSize*kerSize;
    if(xIndex<(imgH-halfKer-1) && yIndex<(imgW-halfKer-1) && xIndex>halfKer+1 && yIndex>halfKer+1)
    {
        float sorted[KERSIZE*KERSIZE];
        int initCnt = 0;
        //store vals
        for(int i=xIndex-halfKer;i<=xIndex+halfKer;i++)
        {
            for(int j=yIndex-halfKer;j<=yIndex+halfKer;j++)
            {
                sorted[initCnt] = inputMat[i*imgW + j];
                initCnt+=1;
            }
             
        }
        
        //sort vals
        for(int i=0;i<cnt;i++)
        {
            for(int j=0;j<cnt;j++)
            {
                if(sorted[j]>sorted[j+1])
                {
                    float temp = sorted[j];
                    sorted[j] = sorted[j+1];
                    sorted[j+1] = temp;
                }
            }
        }

        //replace value
        outputMat[xIndex*imgW + yIndex] = sorted[cnt/2];

    }
}

__global__ void bilinearInterpolation(float* input,float *output, int input_width, int input_height, int output_width, int output_height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= output_width || y >= output_height) return;

    // Compute the coordinates in the input image
    float x_ratio = (float)(input_width - 1) / output_width;
    float y_ratio = (float)(input_height - 1) / output_height;
    float x_input = x * x_ratio;
    float y_input = y * y_ratio;

    int x_low = floor(x_input);
    int y_low = floor(y_input);
    int x_high = min(x_low + 1, input_width - 1);
    int y_high = min(y_low + 1, input_height - 1);

    // Compute the weights for interpolation
    float x_weight = x_input - x_low;
    float y_weight = y_input - y_low;

    // Get the pixel values at the four nearest neighbors
    float top_left = input[y_low * input_width + x_low];
    float top_right = input[y_low * input_width + x_high];
    float bottom_left = input[y_high * input_width + x_low];
    float bottom_right = input[y_high * input_width + x_high];

    // Perform bilinear interpolation
    float top = top_left + (top_right - top_left) * x_weight;
    float bottom = bottom_left + (bottom_right - bottom_left) * x_weight;
    float pixel_value = top + (bottom - top) * y_weight;

    // Write the interpolated value to the output image
    output[y * output_width + x] = pixel_value;
}

__global__ void erosionFilter(float* inputMat,float *outputMat,int imgW,int imgH,int kerSize,int replaceIdx)
{
    int xIndex = blockIdx.x * blockDim.x + threadIdx.x; //image x index or column number
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y; //image y index of row number
    int halfKer = kerSize/2;
    int cnt = kerSize*kerSize;
    if(xIndex<(imgH-halfKer-1) && yIndex<(imgW-halfKer-1) && xIndex>halfKer+1 && yIndex>halfKer+1)
    {
        float sorted[KERSIZE*KERSIZE];
        int initCnt = 0;
        //store vals
        for(int i=xIndex-halfKer;i<=xIndex+halfKer;i++)
        {
            for(int j=yIndex-halfKer;j<=yIndex+halfKer;j++)
            {
                sorted[initCnt] = inputMat[i*imgW + j];
                initCnt+=1;
            }
             
        }
        
        //sort vals
        for(int i=0;i<cnt;i++)
        {
            for(int j=0;j<cnt;j++)
            {
                if(sorted[j]>sorted[j+1])
                {
                    float temp = sorted[j];
                    sorted[j] = sorted[j+1];
                    sorted[j+1] = temp;
                }
            }
        }

        //replace value
        outputMat[xIndex*imgW + yIndex] = sorted[replaceIdx];

    }
}

//////////////////////////////////////////
//
//Caller functions
//
/////////////////////////////////////////
void allocUnifiedMem(int imgW,int imgH)
{
    imgHit = imgH;
    imgWid = imgW;
    szMat = imgH*imgW;
    block1.x = 32;
    block1.y = 32;
    grid1.x = (imgH + block1.x-1)/block1.x;
    grid1.y = (imgW + block1.y-1)/block1.y;

    blockVox.x = 10;
    blockVox.y = 10;
    blockVox.z = 10;
    gridVox.x = (1000 + blockVox.x-1)/blockVox.x;
    gridVox.y = (1000 + blockVox.y-1)/blockVox.y;
    gridVox.z = (1000 + blockVox.z-1)/blockVox.z;



    hipMallocManaged(&inputMat,szMat*sizeof(float));
    hipMallocManaged(&outputMat,szMat*sizeof(float));
    hipMallocManaged(&voxelGrid,VOXSIZE*VOXSIZE*VOXSIZE*sizeof(char));
    hipMallocManaged(&transform,16*sizeof(float));
}




void setUnifiedMem(float* inputMap)
{
    std::memcpy(inputMat,inputMap,szMat*sizeof(float));
}
void setImageBuffs(float* bgd,float* fgd)
{
    std::memcpy(outputMat,bgd,szMat*sizeof(float));
    std::memcpy(inputMat,fgd,szMat*sizeof(float));
}
void destroyUnifiedMem()
{
    hipFree(inputMat);
    hipFree(outputMat);
    hipFree(voxelGrid);
    hipFree(transform);
}

void displaySystemInfo()
{
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, 0);
    std::cout<<"using "<<properties.multiProcessorCount<<" multiprocessors"<<std::endl;
    std::cout<<"max threads per processor: "<<properties.maxThreadsPerMultiProcessor<<std::endl;
}
void toSurfaceCallerUnified()
{   
    // hipError_t err = hipSuccess;
    kernelToSurfaceUnified<<<grid1,block1>>>(inputMat,voxelGrid,outputMat,imgWid,imgHit);
    
    hipDeviceSynchronize();

}

void surfaceToImgCaller(short* depthImg)
{
    for(int xIndex=0;xIndex<VOXSIZE;xIndex++)
    {
        for(int yIndex=0;yIndex<VOXSIZE;yIndex++)
        {
            for(int zIndex=0;zIndex<VOXSIZE;zIndex++)
            {   
                
                if(voxelGrid[xIndex + yIndex*VOXSIZE + zIndex*VOXSIZE*VOXSIZE] ==(char)1)
                {
                    float z = ((float)zIndex*ZDIV)*1000;
                    
                    float xCoor = ((float)(xIndex-500)*XYDIV);
                    float yCoor = ((float)(yIndex-500)*XYDIV);
                    int u = round(((xCoor)*FXY)/z + CX);
                    int v = round(((yCoor)*FXY)/z + CY);
                    if(u<imgHit && v<imgWid )
                    {
                        // printf("%f and %f\n",z,outputMat[u*imgWid+v]);
                        if(outputMat[u*imgWid+v]==0.0 || outputMat[u*imgWid+v]>z)
                        {
                            outputMat[u*imgWid+v] =z;
                            depthImg[u*imgWid+v] = std::floor(outputMat[u*imgWid+v]);

                        }
                    }
                }
            }
        }
    }
    hipDeviceSynchronize();
}
void copyDepthImg(short* depthImg,FilterType T)
{
    if(T == ERODE)
    {
        erosionFilter<<<grid1,block1>>>(outputMat,inputMat,imgWid,imgHit,KERSIZE,KERSIZE*KERSIZE-1);
        // erosionFilter<<<grid1,block1>>>(inputMat,outputMat,imgWid,imgHit,KERSIZE,1);
        hipDeviceSynchronize();
        for(int i=0;i<szMat;i++)
        {
            
            depthImg[i] = std::floor(inputMat[i]);
            if(depthImg[i]==0) depthImg[i] = USMAX;
            
        }
    }
    else if(T == DILATE)
    {
        erosionFilter<<<grid1,block1>>>(outputMat,inputMat,imgWid,imgHit,KERSIZE,1);
        hipDeviceSynchronize();
        for(int i=0;i<szMat;i++)
        {
            
            depthImg[i] = std::floor(inputMat[i]);
            
        }
    }
    else if(T == BILINEAR)
    {
        bilinearInterpolation<<<grid1,block1>>>(outputMat,inputMat,imgWid,imgHit,imgWid,imgHit);
        hipDeviceSynchronize();
        for(int i=0;i<szMat;i++)
        {
            
            depthImg[i] = std::floor(inputMat[i]);
            
        }
    }
    else
    {
        // medFilter<<<grid1,block1>>>(outputMat,inputMat,imgWid,imgHit,KERSIZE);
        hipDeviceSynchronize();
        for(int i=0;i<szMat;i++)
        {
            
            depthImg[i] = std::floor(outputMat[i]);
            
        }
    }
    
}

void combineDepth(short* depthImg,float depthCutoff)
{
    combineDepthKernel<<<grid1,block1>>>(inputMat,outputMat,imgWid,imgHit,depthCutoff);
    medFilter<<<grid1,block1>>>(inputMat,outputMat,imgWid,imgHit,KERSIZE);
    hipDeviceSynchronize();
    for(int i=0;i<szMat;i++)
    {
        
        depthImg[i] = std::floor(outputMat[i]*1000.0f);
        
    }
}
void resetImgSize(int imgW,int imgH)
{
    imgHit = imgH;
    imgWid = imgW;
    szMat = imgH*imgW;
    block1.x = 32;
    block1.y = 32;
    grid1.x = (imgH + block1.x-1)/block1.x;
    grid1.y = (imgW + block1.y-1)/block1.y;
}